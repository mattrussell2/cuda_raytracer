#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <glm/gtc/type_ptr.hpp>
#include <algorithm>

#include "cuda_wrapper_definition.h"
#include "Camera.h"
#include "cone.h"
#include "cube.h"
#include "cylinder.h"
#include "sphere.h"
#include "matrix_multiply.h"
#include "node.h"
#include "screen.h"


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		printf("error");
		return;
		//exit(99);
	}
};

__device__ bool check_light_is_closest(glm::vec3 origin, glm::vec3 ray, float light_t, LightType type, const primitive *d_primitiveList, int numPrimitives) {
	int currShape;
	float t = -1.0;
	origin = origin + ray * 2.0f * 1e-4f;
	for (int i = 0; i < numPrimitives;i++) {
		currShape = d_primitiveList[i].type;
		switch (currShape) {
		case SHAPE_SPHERE:			
			t = sphere::hit(origin, ray, d_primitiveList[i].inverse_transform);
			break;
		case SHAPE_CUBE:
			t = cube::hit(origin, ray, d_primitiveList[i].inverse_transform);
			break;
		case SHAPE_CYLINDER:
			t = cylinder::hit(origin, ray, d_primitiveList[i].inverse_transform);
			break;
		case SHAPE_CONE:
			t = cone::hit(origin, ray, d_primitiveList[i].inverse_transform);
			break;
		default:
			break;
		}
		if (type == LIGHT_DIRECTIONAL && t > 0) return false;
		else if (t > 0 && t < light_t) return false;
	}
	return true;
};

__device__ glm::vec3 calcPhongLight(glm::vec3 eye, glm::vec3 ray, float t, glm::vec3 intersection, glm::vec3 normal, primitive *n, const primitive *d_primitiveList, int d_numPrimitives,
									const SceneLightData *d_lightList, int d_numLights, SceneGlobalData SGD) {
		
	glm::vec3 lineOfSight = glm::normalize(-1.0f * (float)t * ray);	
	
	//start final color as the ambient light for each r,g,b
	glm::vec3 final_color = SGD.ka * n->ambient;	
	glm::vec3 light_ray;
	for (int light = 0; light < d_numLights; light++) {
		if (d_lightList[light].type == LIGHT_DIRECTIONAL) { light_ray = -1.0f * d_lightList[light].dir; }
		else if (d_lightList[light].type == LIGHT_POINT) { light_ray = d_lightList[light].pos - intersection; }
		else continue;

		glm::vec3 worldLightVec(glm::normalize(light_ray));

		//skip this light if the object is closer	
		if (!check_light_is_closest(intersection, worldLightVec, glm::length(light_ray), d_lightList[light].type, d_primitiveList, d_numPrimitives))
			continue;

		float WLVdotNORM = glm::dot(worldLightVec, normal);		
		glm::vec3 reflectedRay = glm::normalize(2.0f * WLVdotNORM * normal - worldLightVec);		
		float REFdotLOS = glm::dot(reflectedRay, lineOfSight);
		if (WLVdotNORM < 0) WLVdotNORM = 0.0f;
		if (REFdotLOS < 0) REFdotLOS = 0.0f;		

		final_color += d_lightList[light].col * (SGD.kd * n->color * WLVdotNORM) + SGD.ks * n->specular * pow(REFdotLOS, n->shininess);		
	}

	return final_color;
};


__device__ void getIntersection(glm::vec3 &eye, glm::vec3 ray, float &t, glm::vec3 &currIntersection, glm::vec3 &currNormal, primitive *&currPrimitive, primitive *d_primitiveList, int d_numPrimitives) {
	t = 100000000;
	float temp_t;	
	primitive *n;

	eye = eye + ray * 2.0f * 1e-4f;	
	for (int i = 0;i < d_numPrimitives;i++) {
		n = &(d_primitiveList[i]);
		switch (n->type) {
		case SHAPE_SPHERE:			
			temp_t = sphere::hit(eye, ray, n->inverse_transform);
			if (temp_t > t || temp_t < 0) continue;
			t = temp_t;
			currNormal = sphere::get_norm(eye, ray, t, n->inverse_transform);
			currIntersection = eye + (float)t * ray;
			currPrimitive = n;
			break;
		case SHAPE_CYLINDER:
			temp_t = cylinder::hit(eye, ray, n->inverse_transform);
			if (temp_t > t || temp_t < 0) continue;
			t = temp_t;
			currNormal = cylinder::get_norm(eye, ray, t, n->inverse_transform);
			currIntersection = eye + (float)t * ray;
			currPrimitive = n;
			break;
		case SHAPE_CUBE:
			temp_t = cube::hit(eye, ray, n->inverse_transform);
			if (temp_t > t || temp_t < 0) continue;
			t = temp_t;
			currNormal = cube::get_norm(eye, ray, t, n->inverse_transform);
			currIntersection = eye + (float)t * ray;
			currPrimitive = n;
			break;
		case SHAPE_CONE:
			temp_t = cone::hit(eye, ray, n->inverse_transform);
			if (temp_t > t || temp_t < 0) continue;
			t = temp_t;
			currNormal = cone::get_norm(eye, ray, t, n->inverse_transform);
			currIntersection = eye + (float)t * ray;
			currPrimitive = n;
			break;
		default:
			break;
		}
	}	
	if (IN_RANGE(t, 100000000)) t = -1.0f;
}

//calculate all the rays and their colors in parallel. 
__global__ void render(glm::vec3 *d_finalColors, int max_x, int max_y,  
					   glm::vec3 *rOrigins, glm::vec3 *rRays, glm::vec3 *rNormals, glm::vec3 *rIntersections, float *rTvals, primitive **rPrimitives,
					   glm::vec3 eyePoint, glm::mat4 inverseCameraTransformMatrix,
					   primitive *d_primitiveList, int d_numPrimitives, const SceneLightData *d_lightList, int d_numLights, SceneGlobalData SGD, 
					   int recursionDepth) {

	// final_color;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	
	//map a pixel to the film plane
	float u = 2.0 * float(i) / float(max_x) - 1.0;
	float v = 2.0 * float(j) / float(max_y) - 1.0;
	float z = -1.0;
	glm::vec3 world_space_pt = glm::vec3(inverseCameraTransformMatrix * glm::vec4(u, v, z, 1.0f));
	
	int start = pixel_index * recursionDepth;	
	glm::vec3 intersection, normal, inboundVector, reflectiveRay;
	float t;
	primitive *p;
	rOrigins[start] = eyePoint;
	rRays[start] = glm::normalize(world_space_pt - eyePoint);
	for (int i = 0; i < recursionDepth; i++) {		
		getIntersection(rOrigins[start + i],rRays[start + i], t, intersection, normal, p, d_primitiveList, d_numPrimitives);
		rTvals[start + i] = t;                                   //t value of the current ray
		if (t < 0) break;                                        //base case (didn't hit anything)	
		rNormals[start + i] = normal;                            //normal of the 'future' intersection
		rIntersections[start + i] = intersection;                //intersection at the point you're headed to
		rPrimitives[start + i] = p;                              //primitive you're GOING to hit
		
		//determine the next ray
		inboundVector = glm::normalize(intersection - rOrigins[start+i]);
		reflectiveRay = glm::normalize(inboundVector - 2.0f * glm::dot(inboundVector, normal) * normal);
		if (i < recursionDepth - 1) {
			rOrigins[start + i + 1] = intersection;                       //save these values for the next 'recursive' call
			rRays[start + i + 1] = reflectiveRay;
		}
	}	

	//now we've got a structure with information for each step of the way. work from the back to front and add up the lighting info.
	glm::vec3 final_color(0.0f); 	
	glm::vec3 local_color;			
	for (int i = recursionDepth - 1; i >= 0; i--) {	
		if (rTvals[start + i] < 0) continue;		
		local_color = calcPhongLight(rOrigins[start + i], rRays[start + i], rTvals[start+i], rIntersections[start+i], rNormals[start+i], rPrimitives[start+i],
									 d_primitiveList, d_numPrimitives, d_lightList, d_numLights, SGD);
		final_color = local_color + SGD.ks * rPrimitives[start + i]->reflective * final_color;					
	}	
	d_finalColors[pixel_index] = final_color;
};


glm::vec3* runCuda(Camera *camera, primitive *primitiveList, int numPrimitives, 
				   SceneLightData *lightList, int numLights, SceneGlobalData SGD) { 

	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipDeviceReset());
	
	int nx = screenWidth;
	int ny = screenHeight;
	int tx = 8;
	int ty = 8;

	int numPixels = screenWidth * screenHeight;
	
	primitive *d_primitiveList; 
	checkCudaErrors(hipMallocManaged((void **)&d_primitiveList, sizeof(primitive) * numPrimitives));
	checkCudaErrors(hipMemcpy(d_primitiveList, primitiveList, sizeof(primitive) * numPrimitives, hipMemcpyHostToDevice));
	
	SceneLightData *d_lightList;
	checkCudaErrors(hipMallocManaged((void **)&d_lightList, sizeof(SceneLightData) * numLights));
	checkCudaErrors(hipMemcpy(d_lightList, lightList, sizeof(SceneLightData) * numLights, hipMemcpyHostToDevice));
		
	int recursionDepth = 5;
	
	glm::vec3 *rOrigins, *rRays, *rNormals, *rIntersections;
	float *rTvals; primitive **rPrimitives;
	
	hipMalloc((void **)&rOrigins, sizeof(glm::vec3) * numPixels * recursionDepth);
	hipMalloc((void **)&rRays, sizeof(glm::vec3) * numPixels * recursionDepth);
	hipMalloc((void **)&rNormals, sizeof(glm::vec3) * numPixels * recursionDepth);
	hipMalloc((void **)&rIntersections, sizeof(glm::vec3) * numPixels * recursionDepth);
	hipMalloc((void **)&rTvals, sizeof(float) * numPixels * recursionDepth);	
	hipMalloc((void **)&rPrimitives, sizeof(primitive*) * numPixels * recursionDepth);
		
	float *rTvalsTmp = new float[numPixels * recursionDepth];
	for (int i = 0;i < numPixels * recursionDepth;i++) {
		rTvalsTmp[i] = -1.0f;
	}
	hipMemcpy(rTvals, rTvalsTmp, sizeof(float) * numPixels * recursionDepth, hipMemcpyHostToDevice);
	delete []rTvalsTmp;

	glm::vec3 *d_finalColors;
	hipMalloc((void **)&d_finalColors, sizeof(glm::vec3) * numPixels);
	
	//get camera information		
	glm::vec3 eye_point = camera->getEyePoint();
	glm::mat4 inverse_transform = camera->getInverseModelViewMatrix() * camera->getInverseScaleMatrix();
	
	clock_t start, stop;
	start = clock();
	
	//number of blocks and threads needed for per-pixel optimization
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	render <<< blocks, threads >>> (d_finalColors, nx, ny, rOrigins, rRays, rNormals, rIntersections, rTvals, rPrimitives,
								    eye_point, inverse_transform, d_primitiveList, numPrimitives, d_lightList, numLights, SGD, recursionDepth);
		
	checkCudaErrors(hipDeviceSynchronize());
		
	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	printf("took %f seconds\n", (float)timer_seconds);

	//copy device memory back to the host --- [the memory is freed MyGLCanvas]
	glm::vec3* h_finalColors = (glm::vec3 *)malloc(sizeof(glm::vec3) * numPixels);
	checkCudaErrors(hipMemcpy(h_finalColors, d_finalColors, sizeof(glm::vec3) * numPixels, hipMemcpyDeviceToHost));
		
	checkCudaErrors(hipFree(rRays));
	checkCudaErrors(hipFree(rNormals));
	checkCudaErrors(hipFree(rIntersections));
	checkCudaErrors(hipFree(rTvals));
	checkCudaErrors(hipFree(rPrimitives));
	checkCudaErrors(hipFree(d_finalColors));	
	checkCudaErrors(hipFree(d_primitiveList));
	checkCudaErrors(hipFree(d_lightList));	
	return h_finalColors;
}